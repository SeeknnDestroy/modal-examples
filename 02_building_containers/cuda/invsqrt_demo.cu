#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Forward declaration of the CUDA kernel
__global__ void invsqrt_kernel(float *out, const float *X, int size);

int main()
{
    int size = 1024;
    float *h_X = (float *)malloc(size * sizeof(float));
    float *h_out = (float *)malloc(size * sizeof(float));
    float *d_X, *d_out;

    // initialize input array with perfect squares
    for (int i = 0; i < size; i++)
    {
        h_X[i] = (float)((i + 1) * (i + 1));
    }

    hipMalloc((void **)&d_X, size * sizeof(float));
    hipMalloc((void **)&d_out, size * sizeof(float));

    hipMemcpy(d_X, h_X, size * sizeof(float), hipMemcpyHostToDevice);

    // Launch the CUDA kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    invsqrt_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_X, size);

    hipMemcpy(h_out, d_out, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print a few of the results
    for (int i = 0; i < 10; i++)
    {
        printf("invsqrt(%f) = %f\n", h_X[i], h_out[i]);
    }

    // Cleanup
    free(h_X);
    free(h_out);
    hipFree(d_X);
    hipFree(d_out);

    return 0;
}
